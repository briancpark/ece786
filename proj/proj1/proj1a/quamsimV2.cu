#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>
#include <vector>

using namespace std;

__global__ void quantum_simulation_gpu(float* U, float* a, float* output, int qubit, int N) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    size_t qid = 1 << qubit;

    if (tid > N)
        return;

    if (tid & qid)
        output[tid] = U[2] * a[tid - qid] + U[3] * a[tid];
    else
        output[tid] = U[0] * a[tid] + U[1] * a[tid + qid];
    __syncthreads();
}

int main(int argc, char** argv) {
    // Parse the command line arguments
    if (argc != 2) {
        fprintf(stderr, "Usage: %s input.txt\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    // Read the input file
    ifstream input_file;
    input_file.open(argv[1]);

    // Setup variables to store matrix and vector
    float *U, *input, *output;
    vector<float> a;
    size_t qubit;
    string line;

    hipMallocManaged(&U, 4 * sizeof(float));

    for (int i = 0; i < 4; i++) {
        input_file >> U[i];
    }

    // Read in the vector until we hit an empty line
    std::getline(input_file, line);
    std::getline(input_file, line);

    while (std::getline(input_file, line) && !line.empty()) {
        a.push_back(stof(line));
    }

    hipMallocManaged(&input, a.size() * sizeof(float));
    for (int i = 0; i < a.size(); i++) {
        input[i] = a[i];
    }

    // Read in the qubit
    input_file >> qubit;

    hipMallocManaged(&output, a.size() * sizeof(float));

    hipDeviceSynchronize();
    int threadsPerBlock = 256;
    int blocksPerGrid = (a.size() + threadsPerBlock - 1) / threadsPerBlock;
#ifdef BENCHMARK
    cudaEvent_t start, stop;
    cudaEventCreate(&start);
    cudaEventCreate(&stop);

    cudaEventRecord(start);
#endif
    quantum_simulation_gpu<<<blocksPerGrid, threadsPerBlock>>>(U, input, output, qubit, a.size());
#ifdef BENCHMARK
    cudaEventRecord(stop);

    cudaEventSynchronize(stop);
    float milliseconds = 0;
    cudaEventElapsedTime(&milliseconds, start, stop);
    cout << "Time taken: " << milliseconds << " ms" << endl;
#endif

    hipDeviceSynchronize();
    // Print the output vector
    for (int i = 0; i < a.size(); i++) {
        printf("%.3f\n", output[i]);
    }

    hipFree(U);
    hipFree(input);
    hipFree(output);

    return 0;
}