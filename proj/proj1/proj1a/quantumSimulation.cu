/*
Develop two versions of the host code, one uses cudaMalloc and cudaMemcpy to move data explicitly
and the other uses cudaMallocManaged to leverage unified virtual memory to move the data. Report the
timing results for both versions.

Input format:

- Your implementation should not be based on specific sizes of the input vector although you can
assume that the vector length is smaller than 230.

- The submitted code will be tested using randomly generated data inputs.

- Example input file:

input file format

- The corresponding output file:

output file format

- The element type should be assumed as the single-precision float.

- In the input file, the first 2x2 matrix represents a single-qubit quantum gate, and the second
128x1 matrix (i.e., vector) represents a 7-qubit (from qubit 0 to qubit 6) quantum state. They are
separated by a blank line. Each matrix row is in a separate line (i.e. ends with a linefeed "\n").
Matrix elements in the same row are separated by a single space. And the number in the last line of
the input  file represents which qubit the single-qubit gate is applied on.

- In the corresponding output format file, the vector represents the 7-qubit quantum output state
after applying the single-qubit gate on qubit 2.



The Expected Output:

- You should output a vector of length N = 2n and print it to the console screen (using stdio).

- Every row should be in a separate line (i.e. it ends with a newline "\n"). The values should be
printed with 3 decimal points precision (not more nor less).
- Other than the output vector, NOTHING else should be printed. Do not print "done". Do not print
the execution time. Do not print anything other than the output vector.

- Note that we will use diff to check the output. That means you need to exactly match the output.
- For grading, your program will be compiled and run with command like "./quamsim ./input.txt"


*/

#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>
#include <vector>

using namespace std;

void read_vector(ifstream input, vector<float>& a) {
    // continuously read each line of the file until we hit an empty line
    // float temp;
    // char* c = "\n";
    // cout << fgetc(fp) << endl;
    // while (fscanf(fp, "%f", &temp) != EOF) {
    //     a.push_back(temp);
    //     // if next line is newline, then break
    //     if (fgetc(fp) == *c) {
    //         break;
    //     }
    //     cout << "added to vector" << temp << endl;
    // }
    // cout << "DONE" << endl;
    return;
}

// void read_qubit(FILE* fp, size_t* qubit) {
//     // fscanf(fp, "%zu", qubit);
//     return;
// }

int main(int argc, char** argv) {
    // Parse the command line arguments
    if (argc != 2) {
        fprintf(stderr, "Usage: %s input.txt\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    // Read the input file
    ifstream input_file;
    input_file.open(argv[1]);

    // Read the first matrix
    // TODO: change to GPU later
    // We know that the matrix is 2x2 guaranteed
    // But for vector, we need to read the file first
    float* U = (float*)malloc(4 * sizeof(float));
    vector<float> a;
    size_t qubit;

    for (int i = 0; i < 4; i++) {
        input_file >> U[i];
    }

    // Read in the vector until we hit an empty line
    // float temp;
    string line;
    std::getline(input_file, line);
    std::getline(input_file, line);

    while (std::getline(input_file, line) && !line.empty()) {
        a.push_back(stof(line));
    }

    // Read in the qubit
    input_file >> qubit;

    // for (auto i : a) {
    //     cout << i << endl;
    // }
    // cout << qubit << endl;
    float* output = (float*)malloc(a.size() * sizeof(float));

    // Perform quantum simulation on qubit
    for (size_t i = 0; i < a.size(); i++) {
        if ((i & (1 << qubit)) == 0) {
            output[i] = U[0] * a[i] + U[1] * a[i + (1 << qubit)];
        } else {
            output[i] = U[2] * a[i - (1 << qubit)] + U[3] * a[i];
        }
    }

    // Print the output vector
    for (int i = 0; i < a.size(); i++) {
        printf("%.3f\n", output[i]);
    }
    return 0;
}