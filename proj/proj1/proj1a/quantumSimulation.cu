/*
Develop two versions of the host code, one uses cudaMalloc and cudaMemcpy to move data explicitly
and the other uses cudaMallocManaged to leverage unified virtual memory to move the data. Report the
timing results for both versions.

Input format:

- Your implementation should not be based on specific sizes of the input vector although you can
assume that the vector length is smaller than 230.

- The submitted code will be tested using randomly generated data inputs.

- Example input file:

input file format

- The corresponding output file:

output file format

- The element type should be assumed as the single-precision float.

- In the input file, the first 2x2 matrix represents a single-qubit quantum gate, and the second
128x1 matrix (i.e., vector) represents a 7-qubit (from qubit 0 to qubit 6) quantum state. They are
separated by a blank line. Each matrix row is in a separate line (i.e. ends with a linefeed "\n").
Matrix elements in the same row are separated by a single space. And the number in the last line of
the input  file represents which qubit the single-qubit gate is applied on.

- In the corresponding output format file, the vector represents the 7-qubit quantum output state
after applying the single-qubit gate on qubit 2.



The Expected Output:

- You should output a vector of length N = 2n and print it to the console screen (using stdio).

- Every row should be in a separate line (i.e. it ends with a newline "\n"). The values should be
printed with 3 decimal points precision (not more nor less).
- Other than the output vector, NOTHING else should be printed. Do not print "done". Do not print
the execution time. Do not print anything other than the output vector.

- Note that we will use diff to check the output. That means you need to exactly match the output.
- For grading, your program will be compiled and run with command like "./quamsim ./input.txt"


*/

#include <iostream>
#include <stdio.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

using namespace std;

void read_matrix(FILE* fp, float* matrix) {
    for (int i = 0; i < 4; i++) {
        fscanf(fp, "%f", &matrix[i]);
    }
}

void read_vector(FILE* fp, float* vector) {
    for (int i = 0; i < 128; i++) {
        fscanf(fp, "%f", &vector[i]);
    }
}

void read_qubit(FILE* fp, size_t* qubit) { fscanf(fp, "%zu", qubit); }

int main(int argc, char** argv) {
    // Parse the command line arguments
    if (argc != 2) {
        fprintf(stderr, "Usage: %s input.txt\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    // Read the input file
    FILE* fp = fopen(argv[1], "r");
    if (fp == NULL) {
        fprintf(stderr, "Failed to open file %s\n", argv[1]);
        exit(EXIT_FAILURE);
    }

    // Read the first matrix
    // TODO: change to GPU later
    float* matrix = (float*)malloc(4 * sizeof(float));
    float* vector = (float*)malloc(128 * sizeof(float));
    float* output = (float*)malloc(128 * sizeof(float));
    size_t qubit;

    // for (int i = 0; i < 4; i++) {
    //     cout << matrix1[i] << " ";
    // }
    read_matrix(fp, matrix);
    read_vector(fp, vector);
    read_qubit(fp, &qubit);

    // cout << qubit << endl;

    // Perform quantum simulation on qubit
    for (size_t i = 0; i < 128; i++) {
        if ((i & (1 << qubit)) == 0) {
            output[i] = matrix[0] * vector[i] + matrix[1] * vector[i + (1 << qubit)];
        } else {
            output[i] = matrix[2] * vector[i - (1 << qubit)] + matrix[3] * vector[i];
        }
    }

    // Print the output vector
    for (int i = 0; i < 128; i++) {
        printf("%.3f\n", output[i]);
    }
    return 0;
}