#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>
#include <vector>

using namespace std;

void quantum_simulation_cpu(float* U, float* a, float* output, size_t qubit, size_t N) {
    // Perform quantum simulation on qubit
    for (size_t i = 0; i < N; i++) {
        if ((i & (1 << qubit)) == 0) {
            output[i] = U[0] * a[i] + U[1] * a[i + (1 << qubit)];
        } else {
            output[i] = U[2] * a[i - (1 << qubit)] + U[3] * a[i];
        }
    }
}

__global__ void device_to_device_memcpy(float* a, float* b, int N) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid > N)
        return;

    b[tid] = a[tid];
    __syncthreads();
}

__global__ void quantum_simulation_gpu(const float* U, const float* a, float* output, int qubit,
                                       int N) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    register size_t qid = 1 << qubit;

    if (tid > N)
        return;

    if (tid & qid)
        output[tid] = U[2] * a[tid - qid] + U[3] * a[tid];
    else
        output[tid] = U[0] * a[tid] + U[1] * a[tid + qid];
    __syncthreads();
    __syncwarp();
}

int main(int argc, char** argv) {
    // Parse the command line arguments
    if (argc != 2) {
        fprintf(stderr, "Usage: %s input.txt\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    // Read the input file
    ifstream input_file;
    input_file.open(argv[1]);

    // Setup variables to store matrix and vector
    float* U_0 = (float*)malloc(4 * sizeof(float));
    float* U_1 = (float*)malloc(4 * sizeof(float));
    float* U_2 = (float*)malloc(4 * sizeof(float));
    float* U_3 = (float*)malloc(4 * sizeof(float));
    float* U_4 = (float*)malloc(4 * sizeof(float));
    float* U_5 = (float*)malloc(4 * sizeof(float));
    vector<float> a;
    size_t qubit_0, qubit_1, qubit_2, qubit_3, qubit_4, qubit_5;

    // Create a list of Us
    vector<float*> Us = {U_0, U_1, U_2, U_3, U_4, U_5};

    // For loop to read in the U matrices
    for (int i = 0; i < 6; i++) {
        float* U = Us[i];
        for (int i = 0; i < 4; i++) {
            input_file >> U[i];
        }
    }

    // Read in the vector until we hit an empty line
    string line;
    getline(input_file, line);
    getline(input_file, line);

    while (getline(input_file, line) && !line.empty()) {
        a.push_back(stof(line));
    }

    // Read in the qubit
    input_file >> qubit_0;
    input_file >> qubit_1;
    input_file >> qubit_2;
    input_file >> qubit_3;
    input_file >> qubit_4;
    input_file >> qubit_5;

    float* output = (float*)malloc(a.size() * sizeof(float));

    // Copy memory to GPU
    float* a_gpu;
    hipMalloc(&a_gpu, a.size() * sizeof(float));
    hipMemcpy(a_gpu, a.data(), a.size() * sizeof(float), hipMemcpyHostToDevice);

    float* output_gpu;
    hipMalloc(&output_gpu, a.size() * sizeof(float));

    float *U_0_gpu, *U_1_gpu, *U_2_gpu, *U_3_gpu, *U_4_gpu, *U_5_gpu;
    hipMalloc(&U_0_gpu, 4 * sizeof(float));
    hipMalloc(&U_1_gpu, 4 * sizeof(float));
    hipMalloc(&U_2_gpu, 4 * sizeof(float));
    hipMalloc(&U_3_gpu, 4 * sizeof(float));
    hipMalloc(&U_4_gpu, 4 * sizeof(float));
    hipMalloc(&U_5_gpu, 4 * sizeof(float));

    hipMemcpy(U_0_gpu, U_0, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(U_1_gpu, U_1, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(U_2_gpu, U_2, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(U_3_gpu, U_3, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(U_4_gpu, U_4, 4 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(U_5_gpu, U_5, 4 * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (a.size() + threadsPerBlock - 1) / threadsPerBlock;

    quantum_simulation_gpu<<<blocksPerGrid, threadsPerBlock>>>(U_0_gpu, a_gpu, output_gpu, qubit_0,
                                                               a.size());
    device_to_device_memcpy<<<blocksPerGrid, threadsPerBlock>>>(output_gpu, a_gpu, a.size());
    quantum_simulation_gpu<<<blocksPerGrid, threadsPerBlock>>>(U_1_gpu, a_gpu, output_gpu, qubit_1,
                                                               a.size());
    device_to_device_memcpy<<<blocksPerGrid, threadsPerBlock>>>(output_gpu, a_gpu, a.size());
    quantum_simulation_gpu<<<blocksPerGrid, threadsPerBlock>>>(U_2_gpu, a_gpu, output_gpu, qubit_2,
                                                               a.size());
    device_to_device_memcpy<<<blocksPerGrid, threadsPerBlock>>>(output_gpu, a_gpu, a.size());
    quantum_simulation_gpu<<<blocksPerGrid, threadsPerBlock>>>(U_3_gpu, a_gpu, output_gpu, qubit_3,
                                                               a.size());
    device_to_device_memcpy<<<blocksPerGrid, threadsPerBlock>>>(output_gpu, a_gpu, a.size());
    quantum_simulation_gpu<<<blocksPerGrid, threadsPerBlock>>>(U_4_gpu, a_gpu, output_gpu, qubit_4,
                                                               a.size());
    device_to_device_memcpy<<<blocksPerGrid, threadsPerBlock>>>(output_gpu, a_gpu, a.size());
    quantum_simulation_gpu<<<blocksPerGrid, threadsPerBlock>>>(U_5_gpu, a_gpu, output_gpu, qubit_5,
                                                               a.size());

    hipMemcpy(output, output_gpu, a.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // Print the output vector
    for (int i = 0; i < a.size(); i++) {
        printf("%.3f\n", output[i]);
    }

    hipFree(U_0_gpu);
    hipFree(U_1_gpu);
    hipFree(U_2_gpu);
    hipFree(U_3_gpu);
    hipFree(U_4_gpu);
    hipFree(U_5_gpu);
    free(U_0);
    free(U_1);
    free(U_2);
    free(U_3);
    free(U_4);
    free(U_5);

    hipFree(a_gpu);
    hipFree(output_gpu);
    free(output);
    return 0;
}